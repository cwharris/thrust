#include "hip/hip_runtime.h"
#include <unittest/unittest.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>


template<typename ExecutionPolicy, typename Iterator, typename T, typename Iterator2>
__global__
void reduce_kernel(ExecutionPolicy exec, Iterator first, Iterator last, T init, Iterator2 result)
{
  *result = thrust::reduce(exec, first, last, init);
}


template<typename T, typename ExecutionPolicy>
void TestReduceDevice(ExecutionPolicy exec, const size_t n)
{
  thrust::host_vector<T>   h_data = unittest::random_integers<T>(n);
  thrust::device_vector<T> d_data = h_data;
  
  thrust::device_vector<T> d_result(1);
  
  T init = 13;
  
  T h_result = thrust::reduce(h_data.begin(), h_data.end(), init);
  
  reduce_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), init, d_result.begin());
  hipError_t const err = hipDeviceSynchronize();
  ASSERT_EQUAL(hipSuccess, err);
  
  ASSERT_EQUAL(h_result, d_result[0]);
}


template<typename T>
struct TestReduceDeviceSeq
{
  void operator()(const size_t n)
  {
    TestReduceDevice<T>(thrust::seq, n);
  }
};
VariableUnitTest<TestReduceDeviceSeq, IntegralTypes> TestReduceDeviceSeqInstance;


template<typename T>
struct TestReduceDeviceDevice
{
  void operator()(const size_t n)
  {
    TestReduceDevice<T>(thrust::device, n);
  }
};
VariableUnitTest<TestReduceDeviceDevice, IntegralTypes> TestReduceDeviceDeviceInstance;

struct noncommutative_reducer
{
  int __device__ operator()(int a, int b) {
    return a;
  }
};

void TestReduceCudaStreams()
{
  typedef thrust::device_vector<int> Vector;

  auto count = 1 << 30;
  auto zeros = thrust::make_constant_iterator<int>(0);

  // Vector v(3);
  // v[0] = 1; v[1] = -2; v[2] = 3;

  auto v = Vector(zeros, zeros + count);

  v[0] = 3;

  hipStream_t s;
  hipStreamCreate(&s);

  // // no initializer
  // ASSERT_EQUAL(thrust::reduce(thrust::cuda::par.on(s), v.begin(), v.end()), 2);

  // commutative
  ASSERT_EQUAL(thrust::reduce(thrust::cuda::par.on(s), v.begin(), v.end(), 7, noncommutative_reducer{}), 7);

  // // with initializer
  // ASSERT_EQUAL(thrust::reduce(thrust::cuda::par.on(s), v.begin(), v.end(), 10), 12);

  hipStreamDestroy(s);
}
DECLARE_UNITTEST(TestReduceCudaStreams);

